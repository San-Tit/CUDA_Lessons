
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // �������� �������� �� GPU.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

	//���� ������ ������ �� ������� ������� ���������
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset ������ ���������� ����� ������� ��� ����, ����� 
	// ����������� �������������� � ������������ �������� ������ ������.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// ��������������� ������� ������������� CUDA ��� ������������� �������� ��������.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	//��������� �� ������ � ������ GPU
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipError_t cudaStatus;

    // ����� GPU ��� �������, ����� ������ � �������� � ����������� GPU.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // ��������� GPU ������ ��� ���� �������� (��� �������, ���� ��������)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // ���������� �������� �������� �� ����������� ������ � ������ GPU.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // ������ GPU � ����� ������� �� ������ ������� �������.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // �������� �� ������ ��� �������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize ���� ��������� ������ ����� �������, � ����������
    // ����� ������, ����������� �� ����� �������.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ����������� �������������� ������ �� ������ GPU (device)� ����������� ������ (host).
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	//������������ ������ �� ����������
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
