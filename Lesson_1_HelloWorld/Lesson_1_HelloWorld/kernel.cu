﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Сложение векторов на GPU.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

	//Если запуск прошел не успешно выводим сообщение
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset должен вызываться перед выходом для того, чтобы 
	// инструменты профилирования и отслеживания показали полные данные. 
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// Вспомогательная функция использования CUDA для параллельного сложения векторов.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	//Указатели на данные в памяти GPU
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipError_t cudaStatus;

    // Выбор GPU для запуска, можно менять в системах с несколькими GPU.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Выделение GPU памяти для трех векторов (два входных, один выходной)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Копировние исходных векторов из оперативной памяти в память GPU.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Запуск GPU с одним потоком на каждый элемент вектора.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Проверка на ошибки при запуске
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize ждет окончания работы всехъ потоков, и возвращает
    // любые ошибки, возникающие во время запуска.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Копирование результирующих данных из памяти GPU (device)в оперативную память (host).
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	//Освобождение памяти на видеокарте
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
